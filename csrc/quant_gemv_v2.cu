// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "dispatch_macros.h"
#include "kernels/quant_gemv_traits.cuh"
#include "kernels/quant_gemv_v2.cuh"
#include "util/common.h"
#include "util/math_utils.h"
namespace vptq {

/**
 * @brief Quantized GEMV kernel.
 * @param act The input activations.
 * @param bias The bias.
 * @param indices The indices.
 * @param centroids The codebook for the main vector quantized weights.
 *        Stored in row-major order. Element type: fp16, bf16.
 *        Shape: (num_codebooks, num_centroids, vec_len).
 * @param residual_centroids The residual centroids.
 * @param scale_weights The scale weights.
 * @param scale_bias The scale bias.
 * @param in_features The number of input features.
 * @param out_features The number of output features.
 */
torch::Tensor quant_gemv_v2(
    const torch::Tensor& act,
    const c10::optional<torch::Tensor>& bias,  //
    const torch::Tensor& indices,              //
    const torch::Tensor& centroids,            //
    const c10::optional<torch::Tensor>& residual_indices,
    const c10::optional<torch::Tensor>& residual_centroids,
    const c10::optional<torch::Tensor>& scale_weights,
    const c10::optional<torch::Tensor>& scale_bias,  //
    int64_t out_features) {
  CHECK_INPUT(act);
  CHECK_INPUT(indices);
  CHECK_INPUT(centroids);

  const at::ScalarType dtype = act.scalar_type();
  TORCH_CHECK(
      dtype == at::ScalarType::Half || dtype == at::ScalarType::BFloat16,
      "the activations must be either half-precision (fp16) or bfloat16.");
  TORCH_CHECK(
      centroids.scalar_type() == dtype,
      "the main centroids must be either half-precision (fp16) or bfloat16.");

  TORCH_CHECK_EQ(act.ndimension(), 3);
  TORCH_CHECK_EQ(centroids.ndimension(), 3);

  const int64_t batch = act.size(0);
  const int64_t seq_length = act.size(1);
  const int64_t in_features = act.size(2);

  const int64_t num_codebooks = centroids.size(0);
  const int64_t num_centroids = centroids.size(1);
  const int64_t vec_len = centroids.size(2);

  TORCH_CHECK_LT(batch * seq_length, 16)
      << "In GEMV, the batch size is suggested to be less than 16.";
  TORCH_CHECK_EQ(num_codebooks, 1) << "Only support one codebook.";
  TORCH_CHECK(
      vec_len == 4 || vec_len == 8 || vec_len == 16,
      "Supported vector length in vectorized quantization: 4, 8, or 16.");

  int64_t num_res_centroids = 0;
  if (residual_centroids.has_value()) {
    CHECK_INPUT(residual_centroids.value());
    TORCH_CHECK_EQ(residual_centroids.value().ndimension(), 3);
    TORCH_CHECK_EQ(residual_centroids.value().size(0), 1)
        << "Only support one codebook.";
    TORCH_CHECK_EQ(residual_centroids.value().size(2), vec_len)
        << "The vector length of the residual centroids must be the same as "
           "the main centroids.";

    num_res_centroids = residual_centroids.value().size(1);
    // once `residual_centroids` has value, `residual_indices`
    // must have value as well
    CHECK_INPUT(residual_indices.value());
  }

  if (scale_weights.has_value()) {
    CHECK_INPUT(scale_weights.value());
    CHECK_INPUT(scale_bias.value());

    TORCH_CHECK(
        scale_weights.value().scalar_type() == dtype,
        "the scale weights must be either half-precision (fp16) or bfloat16.");
    TORCH_CHECK(
        scale_bias.value().scalar_type() == dtype,
        "the scale bias must be either half-precision (fp16) or bfloat16.");
  }

  if (bias.has_value()) {
    CHECK_INPUT(bias.value());
    TORCH_CHECK(bias.value().scalar_type() == dtype,
                "the bias must be either half-precision (fp16) or bfloat16.");
  }

  torch::Tensor output;
  output = at::empty({batch, seq_length, out_features}, centroids.options());

  auto stream = at::cuda::getCurrentCUDAStream().stream();

  int block_z = divup<int64_t, int64_t, int64_t>(out_features, vec_len);
  dim3 blocks(batch * seq_length, num_codebooks, block_z);

  // FIXME(ying): refine the choice of threads in a thread
  // block. For test at the moment.
  static const int kThreads = 4 * WARP_SIZE;
  dim3 threads(kThreads, 1, 1);

  // TODO(ying): this is hardware dependent. Need to make it
  // adaptive.
  const int kMaxSmemPerBlock = 48 * 1024;

  VPTQ_DISPATCH_TYPES(dtype, [&] {
    VPTQ_DISPATCH_VEC_LENGTH(vec_len, [&] {
      VPTQ_DISPATCH_NUM_CENTROIDS(num_centroids, [&] {
        VPTQ_DISPATCH_RES_NUM_CENTROIDS(num_res_centroids, [&] {
          const DType* residual_centroids_ptr =
              residual_centroids.has_value()
                  ? reinterpret_cast<const DType*>(
                        residual_centroids.value().data_ptr())
                  : nullptr;

          const ResIdType* residual_indices_ptr =
              residual_indices.has_value()
                  ? reinterpret_cast<const ResIdType*>(
                        residual_indices.value().data_ptr())
                  : nullptr;

          const DType* bias_ptr =
              bias.has_value()
                  ? reinterpret_cast<const DType*>(bias.value().data_ptr())
                  : nullptr;

          const DType* scale_weights_ptr =
              scale_weights.has_value() ? reinterpret_cast<const DType*>(
                                              scale_weights.value().data_ptr())
                                        : nullptr;

          const DType* scale_bias_ptr = scale_bias.has_value()
                                            ? reinterpret_cast<const DType*>(
                                                  scale_bias.value().data_ptr())
                                            : nullptr;

          static constexpr int kTileSize = 512;

          // NOTE: IdType and ResIdType are declared in the
          // dispatch macros according to
          using Config =
              kernels::QuantGemvKeTraits<DType, IdType, ResIdType, kThreads,
                                         kTileSize, kVecLen, kNumCentroids,
                                         kNumResCentroids>;
          using SharedStorage = Config::SharedStorage;
          int smem_size = SharedStorage::kSmemSize;

          std::cout << "kIdsPerBank: " << Config::kIdsPerBank << std::endl;

          auto kernel =
              &kernels::ke_quant_gemv_v2<DType, IdType, ResIdType,
                                         Config::SharedStorage, Config>;

          // TODO(ying): Check whether shared memory usage
          // exceeds the hardware limit.
          if (smem_size > kMaxSmemPerBlock) {
            hipFuncSetAttribute(reinterpret_cast<const void*>(
                kernel), hipFuncAttributeMaxDynamicSharedMemorySize, smem_size);
          }

          kernel<<<blocks, threads, smem_size, stream>>>(
              reinterpret_cast<DType*>(output.mutable_data_ptr()),
              reinterpret_cast<const DType*>(act.data_ptr()), bias_ptr,
              indices.data_ptr<IdType>(),
              reinterpret_cast<const DType*>(centroids.data_ptr()),
              residual_indices_ptr, residual_centroids_ptr, scale_weights_ptr,
              scale_bias_ptr, batch, seq_length, in_features, out_features);
        });
      });
    });
  });

  return output;
}
}  // namespace vptq
